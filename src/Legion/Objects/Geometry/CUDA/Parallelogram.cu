#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include <Legion/Objects/cuda_common.hpp>
#include <Legion/Objects/Light/CUDA/Light.hpp>

// TODO: attrs should be in a header which can be included by all clients
rtDeclareVariable( legion::LocalGeometry, lgeom, attribute local_geom, ); 
rtDeclareVariable( optix::Ray,            ray,   rtCurrentRay, );

rtDeclareVariable( float4, plane,    , );
rtDeclareVariable( float3, v1,       , );
rtDeclareVariable( float3, v2,       , );
rtDeclareVariable( float3, U,        , );
rtDeclareVariable( float3, V,        , );
rtDeclareVariable( float3, anchor,   , );
rtDeclareVariable( float,  inv_area, , );

RT_PROGRAM void parallelogramIntersect( int )
{
    const float3 n = make_float3( plane );
    const float dt = optix::dot(ray.direction, n );
    const float t  = ( plane.w - optix::dot(n, ray.origin ) ) / dt;

    if( t > ray.tmin && t < ray.tmax ) {
        const float3 p  = ray.origin + ray.direction * t;
        const float3 vi = p - anchor;
        float a1 = optix::dot(v1, vi);
        if(a1 >= 0 && a1 <= 1){
            float a2 = optix::dot(v2, vi);
            if(a2 >= 0 && a2 <= 1){
                if( rtPotentialIntersection( t ) ) {

                    // Fill in a localgeometry
                    legion::LocalGeometry lg;
                    //lg.position_object  = p;
                    lg.position         = p;
                    lg.geometric_normal = n;
                    lg.shading_normal   = n;
                    lg.texcoord         = make_float2( a1, a2 );
                    lgeom = lg;

                    rtReportIntersection( 0 );
                }
            }
        }
    }
}

RT_PROGRAM void parallelogramBoundingBox( int, float result[6] )
{
  // v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
  const float3 tv1  = v1 / optix::dot( v1, v1 );
  const float3 tv2  = v2 / optix::dot( v2, v2 );
  const float3 p00  = anchor;
  const float3 p01  = anchor + tv1;
  const float3 p10  = anchor + tv2;
  const float3 p11  = anchor + tv1 + tv2;
  const float  area = optix::length( optix::cross(tv1, tv2) );
  
  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( p00, p01 ), fminf( p10, p11 ) );
    aabb->m_max = fmaxf( fmaxf( p00, p01 ), fmaxf( p10, p11 ) );
  } else {
    aabb->invalidate();
  }
}


RT_CALLABLE_PROGRAM
legion::LightSample parallelogramSample( float2 sample_seed, float3 shading_point, float3 shading_normal )
{

     legion::LightSample sample;
     sample.pdf = 0.0f;

     const float3 on_light = anchor + sample_seed.x*U + sample_seed.y*V;

     sample.distance = optix::length( on_light - shading_point );
     sample.w_in     = ( on_light - shading_point ) / sample.distance;
     float cosine    = -optix::dot( shading_point, sample.w_in);
     if ( cosine > 0.0f )
     {
         sample.pdf = inv_area*sample.distance*sample.distance / cosine;
         sample.normal = make_float3( plane );
     }

     return sample;
}


RT_CALLABLE_PROGRAM
float parallelogramPDF( float3 w_in, float3 shading_point )
{
    const float3 n  = make_float3( plane );
    const float  dt = optix::dot( w_in, n );
    const float  t  = ( plane.w - optix::dot( n, shading_point ) ) / dt;

    float pdf = 0.0f;

    // Intersect pgram 
    if( t > 0.0f ) 
    {
        const float3 p  = shading_point + w_in * t;
        const float3 vi = p - anchor;
        const float  a1 = optix::dot(v1, vi);
        if( a1 >= 0.0f && a1 <= 1.0f )
        {
            const float a2 = optix::dot( v2, vi );
            if( a2 >= 0.0f && a2 <= 1.0f )
            {
                double dist   = t; 
                double cosine = -optix::dot( n, w_in );
                if ( cosine > 0.0f )
                {
                    pdf = inv_area*dist*dist / cosine;
                }
            }
        }
    }
    return pdf;
}
