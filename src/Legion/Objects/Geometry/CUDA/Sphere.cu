#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include <Legion/Objects/cuda_common.hpp>
#include <Legion/Common/Math/CUDA/ONB.hpp>
#include <Legion/Common/Math/CUDA/Math.hpp>

rtDeclareVariable( float3, center, , );
rtDeclareVariable( float , radius, , );

rtDeclareVariable( legion::LocalGeometry, local_geom, attribute local_geom, ); 
rtDeclareVariable( optix::Ray,            ray,   rtCurrentRay, );


//-----------------------------------------------------------------------------
//
// Helpers
//
//-----------------------------------------------------------------------------

struct IntersectReporter
{
    __device__ __inline__ bool check_t( float t ) { return rtPotentialIntersection( t ); }
    __device__ __inline__ bool report ( float t, legion::LocalGeometry lg ) 
    { 
        local_geom = lg;
        return rtReportIntersection( t ); 
    }
};


struct SampleReporter
{
    __device__ __inline__ SampleReporter( legion::LocalGeometry& lg ) : local_geom( lg ) {}
    __device__ __inline__ bool check_t( float t )
    { return t > 0.0001f; }

    __device__ __inline__ bool report ( float t, legion::LocalGeometry lg ) 
    { local_geom = lg; return true; }

    legion::LocalGeometry& local_geom;
};


template <typename Reporter>
static __device__ __inline__
bool sphereIntersectImpl( 
        float3 origin,
        float3 direction, 
        float3 center, 
        float  radius,
        Reporter& reporter )
{
    float3 O = origin - center;
    float3 D = direction;

    float b = optix::dot(O, D);
    float c = optix::dot(O, O)-radius*radius;
    float disc = b*b-c;
    
    bool intersection_found = false;

    if(disc > 0.0f)
    {
        float sdisc = sqrtf(disc);
        float root1 = (-b - sdisc);


        float root11 = 0.0f;

        // refine root1
        if( fabsf(root1) > 10.f * radius )
        {
            float3 O1 = O + root1 * direction;
            b = optix::dot(O1, D);
            c = optix::dot(O1, O1) - radius*radius;
            disc = b*b - c;

            if(disc > 0.0f)
            {
                sdisc = sqrtf(disc);
                root11 = (-b - sdisc);
            }
        }

        const float t = root1 + root11;
        if( reporter.check_t( t ) ) 
        {

            const float3 normal = (O + t*D)/radius;

            // Fill in a localgeometry
            legion::LocalGeometry lg;
            lg.position         = origin + t*direction;
            lg.geometric_normal = normal;
            lg.shading_normal   = normal;
            lg.texcoord         = make_float2( 0.0f );

            intersection_found = reporter.report( t, lg );
        } 

        if( !intersection_found )
        {
            const float t = (-b + sdisc) +  root1;
            if( reporter.check_t( t ) ) 
            {
                const float3 normal = (O + t*D)/radius;

                // Fill in a localgeometry
                legion::LocalGeometry lg;
                lg.position         = origin + t*direction;
                lg.geometric_normal = normal;
                lg.shading_normal   = normal;
                lg.texcoord         = make_float2( 0.0f );

                intersection_found = reporter.report( t, lg );
            }
        }
    }

    return intersection_found;
}


//-----------------------------------------------------------------------------
//
//
//
//-----------------------------------------------------------------------------
RT_PROGRAM void sphereIntersect( int )
{
    /*
    // TODO: Bug in optix intersection inlining is breaking this
    IntersectReporter reporter;
    sphereIntersectImpl<IntersectReporter>(
                ray.origin,
                ray.direction,
                center,
                radius,
                reporter );
    */

    float3 O = ray.origin - center;
    float3 D = ray.direction;

    float b = optix::dot(O, D);
    float c = optix::dot(O, O)-radius*radius;
    float disc = b*b-c;
    if(disc > 0.0f)
    {
        float sdisc = sqrtf(disc);
        float root1 = (-b - sdisc);


        float root11 = 0.0f;

        // refine root1
        if( fabsf(root1) > 10.f * radius )
        {
            float3 O1 = O + root1 * ray.direction;
            b = optix::dot(O1, D);
            c = optix::dot(O1, O1) - radius*radius;
            disc = b*b - c;

            if(disc > 0.0f)
            {
                sdisc = sqrtf(disc);
                root11 = (-b - sdisc);
            }
        }

        bool check_second = true;
        const float t = root1 + root11;
        if( rtPotentialIntersection( t ) ) {

            const float3 normal = (O + t*D)/radius;

            // Fill in a localgeometry
            legion::LocalGeometry lg;
            lg.position         = ray.origin + t*ray.direction;
            lg.geometric_normal = normal;
            lg.shading_normal   = normal;
            lg.texcoord         = make_float2( 0.0f );

            local_geom = lg;

            if(rtReportIntersection(0))
                check_second = false;
        } 

        if(check_second) {

            const float t = (-b + sdisc) +  root1;
            if( rtPotentialIntersection( t ) ) {

                const float3 normal = (O + t*D)/radius;

                // Fill in a localgeometry
                legion::LocalGeometry lg;
                lg.position         = ray.origin + t*ray.direction;
                lg.geometric_normal = normal;
                lg.shading_normal   = normal;
                lg.texcoord         = make_float2( 0.0f );

                local_geom = lg;

                rtReportIntersection(0);
            }
        }
    }
}

//-----------------------------------------------------------------------------
//
//
//
//-----------------------------------------------------------------------------

RT_PROGRAM void sphereBoundingBox( int, float result[6] )
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( radius > 0.0f  && !isinf(radius) ) {
    aabb->m_min = center - make_float3( radius );
    aabb->m_max = center + make_float3( radius );
  } else {
    aabb->invalidate();
  }
}

//-----------------------------------------------------------------------------
//
//
//
//-----------------------------------------------------------------------------

RT_CALLABLE_PROGRAM
legion::LightSample sphereSample( float2 sample_seed, float3 shading_point )
{
    legion::LightSample sample;
    sample.pdf = 0.0f;

    float3 temp = center - shading_point;
    float d = optix::length( temp );
    temp /= d;
    
    if ( d <= radius )
        return sample;

    // internal angle of cone surrounding light seen from viewpoint
    float sin_alpha_max = (radius / d);
    float cos_alpha_max = sqrtf( 1.0f - sin_alpha_max*sin_alpha_max );

    float q    = 2.0f*legion::PI*( 1.0f - cos_alpha_max ); // solid angle
    sample.pdf =  1.0f/q;                          // pdf is one / solid angle

    const float phi       = 2.0f*legion::PI*sample_seed.x;
    const float cos_theta = 1.0f - sample_seed.y * ( 1.0f - cos_alpha_max );
    const float sin_theta = sqrtf( 1.0f - cos_theta*cos_theta );
    const float cos_phi = cosf( phi );
    const float sin_phi = sinf( phi );

    legion::ONB uvw( temp );
    float3 w_in = optix::normalize( make_float3( cos_phi*sin_theta, sin_phi*sin_theta, cos_theta) );
    w_in = uvw.inverseTransform( w_in );

    SampleReporter reporter( sample.point_on_light );
    if( !sphereIntersectImpl<SampleReporter>(
                shading_point,
                w_in,
                center,
                radius,
                reporter ) )
        sample.pdf = 0.0f;

    return sample;
}

