#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

/// \file IGeometry.hpp
/// Pure virtual interface for Geometry classes


#include <Legion/Objects/cuda_common.hpp>


rtDeclareVariable( float3, center, , );
rtDeclareVariable( float , radius, , );

// TODO: attrs should be in a header which can be included by all clients
rtDeclareVariable( legion::LocalGeometry, lgeom, attribute local_geom, ); 
rtDeclareVariable( optix::Ray,            ray,   rtCurrentRay, );


RT_PROGRAM void sphereIntersect( int )
{
    // TODO: rename these origin, direction
    float3 O = ray.origin - center;
    float3 D = ray.direction;

    float b = optix::dot(O, D);
    float c = optix::dot(O, O)-radius*radius;
    float disc = b*b-c;
    if(disc > 0.0f)
    {
        float sdisc = sqrtf(disc);
        float root1 = (-b - sdisc);

        bool do_refine = false;

        float root11 = 0.0f;

        // refine root1
        float3 O1 = O + root1 * ray.direction;
        b = optix::dot(O1, D);
        c = optix::dot(O1, O1) - radius*radius;
        disc = b*b - c;

        if(disc > 0.0f) {
            sdisc = sqrtf(disc);
            root11 = (-b - sdisc);
        }

        bool check_second = true;
        if( rtPotentialIntersection( root1 + root11 ) ) {

            const float  t      = root1 + root11;
            const float3 normal = (O + t*D)/radius;

            // Fill in a localgeometry
            legion::LocalGeometry lg;
            lg.position_object  = ray.origin + t*ray.direction;
            lg.geometric_normal = normal;
            lg.shading_normal   = normal;
            lg.texcoord         = make_float2( 0.0f );

            lgeom = lg;

            if(rtReportIntersection(0))
                check_second = false;
        } 

        if(check_second) {
            float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
            if( rtPotentialIntersection( root2 ) ) {
                const float  t      = root2; 
                const float3 normal = (O + t*D)/radius;

                // Fill in a localgeometry
                legion::LocalGeometry lg;
                lg.position_object  = ray.origin + t*ray.direction;
                lg.geometric_normal = normal;
                lg.shading_normal   = normal;
                lg.texcoord         = make_float2( 0.0f );

                lgeom = lg;

                rtReportIntersection(0);
            }
        }
    }
}


RT_PROGRAM void sphereBoundingBox( int, float result[6] )
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( radius > 0.0f  && !isinf(radius) ) {
    aabb->m_min = center - make_float3( radius );
    aabb->m_max = center + make_float3( radius );
  } else {
    aabb->invalidate();
  }
}

