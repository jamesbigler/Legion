#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include <Legion/Objects/cuda_common.hpp>
#include <optixu/optixu_math_namespace.h>


rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable( unsigned, sample_index, , );

rtBuffer<float4, 2> output_buffer;



RT_PROGRAM void progressiveRendererRayGen()
{
    float2 screen_sample;
    float2 lens_sample;
    float  time_sample;
    legion::generateSobolSamples( launch_dim,
                                  launch_index,
                                  sample_index,
                                  screen_sample,
                                  lens_sample,
                                  time_sample );
    
    legion::RayGeometry rg = legionCameraCreateRay( lens_sample,
                                                    screen_sample,
                                                    time_sample );

    legion::RadiancePRD prd;
    prd.result = make_float3( 0.0f );
    prd.importance = 1.0f;
    prd.depth = 0u;

    optix::Ray ray = optix::make_Ray( 
            rg.origin,
            rg.direction,
            0u,
            0.0f,
            RT_DEFAULT_MAX );
    rtTrace( legion_top_group, ray, prd );

    const float4 prev   = output_buffer[ launch_index ];
    const float4 cur    = make_float4( prd.result, 1.0f );
    const float4 result = optix::lerp( prev, cur, 1.0f / static_cast<float>( sample_index+1 ) );
    output_buffer[ launch_index ] = result;
}






