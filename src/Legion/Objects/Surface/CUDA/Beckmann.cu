#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include <Legion/Objects/cuda_common.hpp>
#include <Legion/Objects/Surface/CUDA/Surface.hpp>
#include <Legion/Objects/Surface/CUDA/Microfacet.hpp>
#include <Legion/Objects/Texture/CUDA/Texture.hpp>
#include <Legion/Common/Math/CUDA/ONB.hpp>
#include <Legion/Common/Math/CUDA/Math.hpp>


/*
rtDeclareVariable( float3, reflectance, , );
rtDeclareVariable( float,  alpha, , );
*/

legionDeclareTexture( float4, reflectance );
legionDeclareTexture( float,  alpha       );

using namespace legion;

RT_CALLABLE_PROGRAM
legion::BSDFSample beckmannSampleBSDF( 
        float3 seed,
        float3 w_out,
        legion::LocalGeometry p )
{
    const float4 R = legionTex( reflectance, p, w_out );
    const float  a = legionTex( alpha, p, w_out );

    BeckmannDistribution distribution( a );
    NopFresnel           fresnel;
    MicrofacetSurface<BeckmannDistribution, NopFresnel> 
        surface( make_float3( R ), distribution, fresnel );
    legion::BSDFSample sample = surface.sample( make_float2( seed ), w_out, p );

    CHECK_FINITE( sample.w_in       );
    CHECK_FINITE( sample.f_over_pdf );
    CHECK_FINITE( sample.pdf        );
    return sample;
}


RT_CALLABLE_PROGRAM
float4 beckmannEvaluateBSDF(
        float3 w_out,
        legion::LocalGeometry p,
        float3 w_in )
{
    const float4 R = legionTex( reflectance, p, w_out );
    const float  a = legionTex( alpha, p, w_out );

    BeckmannDistribution distribution( a );
    NopFresnel           fresnel;
    MicrofacetSurface<BeckmannDistribution, NopFresnel> 
        surface( make_float3( R ), distribution, fresnel );
    const float4 val = surface.evaluate( w_out, p, w_in );

    CHECK_FINITE( val );
    return val;
}


RT_CALLABLE_PROGRAM
float beckmannPDF( float3 w_out, legion::LocalGeometry p, float3 w_in )
{
    const float3 R = make_float3( 0.0f ); // Not used in pdf
    const float  a = legionTex( alpha, p, w_out );

    BeckmannDistribution distribution( a );
    NopFresnel           fresnel;
    MicrofacetSurface<BeckmannDistribution, NopFresnel> 
        surface( R , distribution, fresnel );
    const float pdf = surface.pdf( w_out, p, w_in );
    
    CHECK_FINITE( pdf );
    return pdf;
}
