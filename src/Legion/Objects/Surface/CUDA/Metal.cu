#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include <Legion/Objects/cuda_common.hpp>
#include <Legion/Objects/Surface/CUDA/Surface.hpp>
#include <Legion/Objects/Surface/CUDA/Microfacet.hpp>
#include <Legion/Objects/Texture/CUDA/Texture.hpp>
#include <Legion/Common/Math/CUDA/Math.hpp>


legionDeclareTexture( float4, reflectance );
legionDeclareTexture( float,  alpha       );
legionDeclareTexture( float4, eta         );
legionDeclareTexture( float4, k           );

using namespace legion;

RT_CALLABLE_PROGRAM
legion::BSDFSample metalSampleBSDF( 
        float3 seed,
        float3 w_out,
        legion::LocalGeometry p )
{
    const float4 R      = legionTex( reflectance, p.texcoord, p.position );
    const float  alpha_ = legionTex( alpha, p.texcoord, p.position );
    const float4 eta_   = legionTex( eta  , p.texcoord, p.position );
    const float4 k_     = legionTex( k    , p.texcoord, p.position );

    BeckmannDistribution distribution( alpha_ );
    ConductorFresnel     fresnel( make_float3( eta_ ), make_float3( k_ ) );
    MicrofacetSurface<BeckmannDistribution, ConductorFresnel> 
        surface( make_float3( R ), distribution, fresnel );
    return surface.sample( make_float2( seed ), w_out, p );
}


RT_CALLABLE_PROGRAM
float4 metalEvaluateBSDF(
        float3 w_out,
        legion::LocalGeometry p,
        float3 w_in )
{
    const float4 R      = legionTex( reflectance, p.texcoord, p.position );
    const float  alpha_ = legionTex( alpha, p.texcoord, p.position );
    const float4 eta_   = legionTex( eta  , p.texcoord, p.position );
    const float4 k_     = legionTex( k    , p.texcoord, p.position );

    BeckmannDistribution distribution( alpha_ );
    ConductorFresnel     fresnel( make_float3( eta_ ), make_float3( k_ ) );
    MicrofacetSurface<BeckmannDistribution, ConductorFresnel> 
        surface( make_float3( R ), distribution, fresnel );
    return surface.evaluate( w_out, p, w_in );
}


RT_CALLABLE_PROGRAM
float metalPDF( float3 w_out, legion::LocalGeometry p, float3 w_in )
{
    const float3 R = make_float3( 0.0f ); // Not used in pdf
    const float  a = legionTex( alpha, p.texcoord, p.position );

    BeckmannDistribution distribution( a );
    NopFresnel           fresnel;        // Not used in pdf
    MicrofacetSurface<BeckmannDistribution, NopFresnel> 
        surface( R , distribution, fresnel );
    return surface.pdf( w_out, p, w_in );
}
