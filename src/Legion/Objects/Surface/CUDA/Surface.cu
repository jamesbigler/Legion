#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

/// \file IGeometry.hpp
/// Pure virtual interface for Geometry classes


#include <Legion/Objects/cuda_common.hpp>


rtDeclareVariable( legion::LocalGeometry, lgeom,    attribute local_geom, ); 
rtDeclareVariable( optix::Ray,            ray,      rtCurrentRay, );
rtDeclareVariable( float,                 t_hit,    rtIntersectionDistance, );



RT_PROGRAM
void legionAnyHit()
{
    shadow_prd.hit_p = ray.origin + t_hit * ray.direction;
    shadow_prd.occluded = 1u; 
    rtTerminateRay();
}


RT_PROGRAM
void legionClosestHit()
{
    float3 result = make_float3( 0.0f );
    const float3 P = ray.origin + t_hit * ray.direction;
    
    legion::LocalGeometry local_geom = lgeom;
    local_geom.position = P;

    //
    // emitted contribution
    //
    if( radiance_prd.count_emitted_light )
    {
        const float3 w_out = -ray.direction;
        result += legionSurfaceEmission( w_out, local_geom );
    }

    //
    // direct lighting
    //
    const int num_lights = 1;

    for( int i = 0; i < num_lights; ++i )
    {

        float2 seed = make_float2( 0.5f );
        const legion::LightSample light_sample = legionLightSample( seed, P ); 

        float3       w_in       = light_sample.point_on_light.position - P;
        const float  light_dist = optix::length( w_in );
        w_in /= light_dist;

        // occlusion query
        bool occluded = optix::dot( w_in, local_geom.shading_normal ) <= 0.0f;
        if( !occluded )
            occluded = legion::pointOccluded( P, w_in, light_dist );  

        if( !occluded )
        {
            const float3 light_col = 
                legionLightEmission( -w_in, light_sample.point_on_light );
            const float3 w_out     = -ray.direction;
            result += light_col * 
                      legionSurfaceEvaluateBSDF( w_out, local_geom, w_in );
            result =  light_sample.point_on_light.position; 
        }
    }


    //
    // indirect lighting
    //


    radiance_prd.result = result;
}
