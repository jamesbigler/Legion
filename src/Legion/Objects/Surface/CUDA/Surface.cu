#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

/// \file IGeometry.hpp
/// Pure virtual interface for Geometry classes


#include <Legion/Common/Math/CUDA/Math.hpp>
#include <Legion/Common/Math/CUDA/Sobol.hpp>
#include <Legion/Objects/Light/CUDA/Light.hpp>
#include <Legion/Objects/Surface/CUDA/Surface.hpp>
#include <Legion/Objects/cuda_common.hpp>


rtDeclareVariable( legion::LocalGeometry, local_geom, attribute local_geom, ); 
rtDeclareVariable( optix::Ray,            ray,        rtCurrentRay, );
rtDeclareVariable( float,                 t_hit,      rtIntersectionDistance, );



RT_PROGRAM
void legionAnyHit()
{
    shadow_prd.hit_p = ray.origin + t_hit * ray.direction;
    shadow_prd.occluded = 1u; 
    rtTerminateRay();
}


RT_PROGRAM
void legionClosestHit() // MIS
{
    float3 radiance = make_float3( 0.0f );

    //
    // Emitted contribution
    //
    float w                     = 1.0f;
    const float3 w_out          = -ray.direction;
    const float  choose_light_p = 1.0f / static_cast<float>( legionLightCount );
    const float3 P = ray.origin + t_hit * ray.direction;
    if( !radiance_prd.count_emitted_light )
    {
        const float light_pdf = legionLightPDF( ray.direction, P ); 
        w = legion::powerHeuristic( radiance_prd.pdf, light_pdf*choose_light_p );
    }
    radiance  = w * legionSurfaceEmission( w_out, local_geom );

    const unsigned sobol_index = radiance_prd.sobol_index;

    // 
    // Indirect lighting
    //
    const float2 bsdf_seed = 
        make_float2( 
                legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ),
                legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ) );

    legion::BSDFSample bsdf_sample = 
        legionSurfaceSampleBSDF( bsdf_seed, w_out, local_geom );


    const float  cosine   = optix::dot( bsdf_sample.w_in, local_geom.shading_normal ); // TODO: redundant
    radiance_prd.origin              = P;
    radiance_prd.direction           = bsdf_sample.w_in;
    radiance_prd.attenuation         = bsdf_sample.f_over_pdf;// * cosine;
    radiance_prd.done                = false; 
    radiance_prd.pdf                 = bsdf_sample.pdf; 
    radiance_prd.count_emitted_light = false; 

    
    //
    // direct lighting
    //

    // TODO: If not bsdf.isSingular
    const float    choose_light_seed = legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ );
    const unsigned light_index       = choose_light_seed * legionLightCount;

    const float2 light_seed = 
        make_float2( 
                legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ),
                legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ) );

    // TODO fold lightEvaluate into this
    const legion::LightSample light_sample = 
        legion::lightSample( light_index, light_seed, P, local_geom.shading_normal  ); 

    if( light_sample.pdf > 0.0f )
    {
        const float cos_theta = optix::dot( light_sample.w_in, local_geom.shading_normal );
        if( cos_theta > 0.0f ) 
        {
            const float3 w_out    = -ray.direction;
            const float  bsdf_pdf = legionSurfacePDF( w_out, local_geom, light_sample.w_in ); // TODO: fold into evaluate


            if( bsdf_pdf > 0.0f ) // TODO: redundant with above check on dot product
            {

                if( !legion::pointOccluded( P, light_sample.w_in, light_sample.distance ) )
                {
                    const float3 light_radiance = 
                        legion::lightEvaluate( 
                                light_index, 
                                light_sample.w_in, 
                                light_sample.distance,
                                light_sample.normal );
            

                    const float3 w_out  = -ray.direction;
                    const float  weight = legion::powerHeuristic( light_sample.pdf*choose_light_p, bsdf_pdf );
                    const float3 bsdf   = legionSurfaceEvaluateBSDF( w_out, local_geom, light_sample.w_in );


                    radiance +=  light_radiance * bsdf * ( weight  / ( light_sample.pdf*choose_light_p ) );
                }
            }
        }
    }

    //
    // Report result
    // 
    radiance_prd.radiance = radiance;
}


/*
RT_PROGRAM
void legionClosestHit2() // No mis`
{
    float3 radiance = make_float3( 0.0f );

    //
    // Emitted contribution
    //
    if( radiance_prd.count_emitted_light )
    {
        const float3 w_out = -ray.direction;
        radiance  = legionSurfaceEmission( w_out, local_geom );
    }

    // 
    // Indirect lighting
    //
    const unsigned sobol_index = radiance_prd.sobol_index;
    const float2 seed = 
        make_float2( 
            legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ),
            legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ) );

    const float3 w_out = -ray.direction;
    legion::BSDFSample bsdf_sample = 
        legionSurfaceSampleBSDF( seed, w_out, local_geom );

    const float3 P = ray.origin + t_hit * ray.direction;

    radiance_prd.origin              = P;
    radiance_prd.direction           = bsdf_sample.w_in;
    radiance_prd.attenuation         = bsdf_sample.f_over_pdf;
    radiance_prd.done                = false; 
    radiance_prd.count_emitted_light = false; 

    //
    // direct lighting
    //
    const unsigned num_lights  = 1;
    for( unsigned i = 0; i < num_lights; ++i )
    {
        const float2 seed = 
            make_float2( 
                legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ),
                legion::Sobol::gen( sobol_index, radiance_prd.sobol_dim++ ) );

        const legion::LightSample light_sample = legionLightSample( seed, P ); 
        if( light_sample.pdf > 0.0f )
        {
            float3       w_in       = light_sample.point_on_light.position - P;
            const float  light_dist = optix::length( w_in );
            w_in /= light_dist;

            if( optix::dot( w_in, local_geom.shading_normal ) > 0.0f ) 
            {
                if( !legion::pointOccluded( P, w_in, light_dist ) )
                {
                    const float3 light_col = 
                        legionLightEmission( 
                                -w_in, 
                                light_sample.point_on_light );

                    const float3 w_out = -ray.direction;
                    const float3 bsdf = 
                        legionSurfaceEvaluateBSDF( 
                                w_out, 
                                local_geom, 
                                w_in );

                    radiance +=  light_col * bsdf / light_sample.pdf;
                }
            }
        }
    }

    //
    // Report result
    // 
    radiance_prd.radiance = radiance;
}
*/
