#include "hip/hip_runtime.h"

// Copyright (C) 2011 R. Keith Morley 
// 
// (MIT/X11 License)
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to
// deal in the Software without restriction, including without limitation the
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
// sell copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include <optix.h>

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<float4, 2> output_buffer;

RT_PROGRAM void legionCamera()
{
    const float r = static_cast<float>( launch_index.x ) /
                    static_cast<float>( launch_dim.x );
    const float g = static_cast<float>( launch_index.y ) /
                    static_cast<float>( launch_dim.y );


    output_buffer[ launch_index ] = make_float4( r, g, 0.0f, 1.0f );
}






